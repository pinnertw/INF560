#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void
compute_stringKernel(char* res, char* a, char* b, char* c, int length){
/* Add on device with kernel execution */
    int i;
    i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<length){
        res[i] = a[i] + b[i] + c[i] ; 
    }
}

/* Function computing the final string to print */
void compute_string( char * res, char * a, char * b, char *c, int length ) 
{
    int total_size=length*sizeof(char);

    /* Define device variables */
    char * d_res ;
    char * d_a ;
    char * d_b ;
    char * d_c ;

    /* Allocation of memory */
    hipMalloc((void **)&d_res, total_size);
    hipMalloc((void **)&d_a, total_size);
    hipMalloc((void **)&d_b, total_size);
    hipMalloc((void **)&d_c, total_size);

    /* Copy array from CPU to device */
    hipMemcpy(d_res, res, total_size, hipMemcpyHostToDevice);
    hipMemcpy(d_a, a, total_size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, total_size, hipMemcpyHostToDevice);
    hipMemcpy(d_c, c, total_size, hipMemcpyHostToDevice);

    /* execute the kernel */
    compute_stringKernel<<<1, 30>>>(d_res, d_a, d_b, d_c, length);

    /* return the result from device to CPU */
    hipMemcpy(res, d_res, total_size, hipMemcpyDeviceToHost);
}

int main()
{

char * res ;

char a[30] = { 40, 70, 70, 70, 80, 0, 50, 80, 80, 70, 70, 0, 40, 80, 79, 70, 0, 40, 50, 50, 0, 70, 80, 0, 30, 50, 30, 30, 0, 0 } ;
char b[30] = { 10, 10, 10, 10, 10, 10, 10, 10, 10, 10, 10, 10, 10, 10, 10, 
10, 10, 10, 10, 10, 10, 10, 10, 10, 10, 10, 10, 10, 0, 0 } ;
char c[30] = { 22, 21, 28, 28, 21, 22, 27, 21, 24, 28, 20, 22, 20, 24, 22, 
29, 22, 21, 20, 25, 22, 25, 20, 22, 27, 25, 28, 25, 0, 0 } ;

res = (char *)malloc( 30 * sizeof( char ) ) ;


/* This function call should be programmed in CUDA */
/* -> need to allocate and transfer data to/from the device */
compute_string( res, a, b, c, 30 ) ;

printf( "%s\n", res ) ;

return 0 ;
}
