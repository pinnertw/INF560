#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#define DEBUG false

__global__ void
vectoraddKernel(int* A, int* B, int* C, int n){
    int i;
    int index;
    int nb_threads;
    index=blockIdx.x * blockDim.x + threadIdx.x;
    nb_threads=blockDim.x * gridDim.x;
    if (index<n){
        for (i=index; i<n; i+=nb_threads){
            C[i] = A[i]+B[i];
        }
    }
}

void vectoradd(int* A, int* B, int* C, int n){
    int total_size = n*sizeof(int);

    /* Define device variables */
    int* dA;
    int* dB;
    int* dC;

    /* Allocation of memory */
    hipMalloc((void **)&dA, total_size);
    hipMalloc((void **)&dB, total_size);
    hipMalloc((void **)&dC, total_size);

    /* Copy array from CPU to device */
    hipMemcpy(dA, A, total_size, hipMemcpyHostToDevice);
    hipMemcpy(dB, B, total_size, hipMemcpyHostToDevice);

    /* execute the kernel */
    vectoraddKernel<<<1, 100>>>(dA, dB, dC, n);

    /* return the result from device to CPU */
    hipMemcpy(C, dC, total_size, hipMemcpyDeviceToHost);

}

int main(int argc, char**argv) {
  int s ;
  int n ;
  int j;
  struct timeval t1, t2;
  double duration;

  /* Check the input arguments */
  if(argc <3) {
    printf("Usage: %s N S\n", argv[0]);
    printf( "\tN: size of the array\n" ) ;
    printf( "\tS: seed for pseudo-random generator\n" ) ;
    exit( 1 ) ;
  }

  n = atoi(argv[1]);
  s = atoi(argv[2]);
  srand48(s);

  /* Allocate the array */
  int * A;
  int * B;
  int * C;
  A = (int *)malloc(n * sizeof(int));
  B = (int *)malloc(n * sizeof(int));
  C = (int *)malloc(n * sizeof(int));
  if ( A == NULL || B == NULL || C == NULL) { 
      fprintf( stderr, "Unable to allocate %d elements\n", n ) ;
      return 1 ; 
  }

  /* Initialize the array */
  for(j=0; j<n; j++){
      A[j] = lrand48()%n;
      B[j] = lrand48()%n;
  }

  /* start the measurement */
  gettimeofday(&t1, NULL);

  /* Add A, B to C */
  vectoradd(A, B, C, n);

  /* end the measurement */
  gettimeofday(&t2, NULL);
  duration = (t2.tv_sec - t1.tv_sec)+((t2.tv_usec-t1.tv_usec)/1e6);

#if DEBUG
  printf("the array 1 contains:\n");
  for(i=0; i<n; i++) {
    printf("%d  ", A[i]);
  }
  printf("\n");
  printf("the array 2 contains:\n");
  for(i=0; i<n; i++) {
    printf("%d  ", B[i]);
  }
  printf("\n");
  /* print result */
  printf("results:\n");
  for(i=0; i<n; i++) {
    printf("%d  ", C[i]);
  }
  printf("\n");
#endif
  printf("(Seed %d, Size %d) Computation time : %g s\n", s, n, duration);
  return 0;
}
